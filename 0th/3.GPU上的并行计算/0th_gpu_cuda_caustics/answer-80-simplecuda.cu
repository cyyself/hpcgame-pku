
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

struct lens_t {
    float x, y, r, d;
};

struct pane_t {
    int panecount;
    float *panepos;
    int *lensindex;
    lens_t *lensdata;
};

struct result_t {
    int raydensity;
    int sensordensity;
    int *sensordata;
};

int loadconf(const char *fn, pane_t &pane, result_t &result) {
    size_t n;
    FILE *fi;
    if (fi = fopen(fn, "rb")) {
        n = fread(&result.raydensity, 4, 1, fi);
        n = fread(&result.sensordensity, 4, 1, fi);
        // result.sensordata = new std::atomic<int>[result.sensordensity*result.sensordensity];
        hipMallocManaged(&result.sensordata, result.sensordensity*result.sensordensity * sizeof(int));
        hipMemset(&result.sensordata, 0, result.sensordensity*result.sensordensity * sizeof(int));
        n = fread(&pane.panecount, 4, 1, fi);
        // pane.panepos = (float *)malloc(pane.panecount * sizeof(float));
        hipMallocManaged(&pane.panepos, pane.panecount * sizeof(float));
        n = fread(pane.panepos, 4, pane.panecount, fi);
        // pane.lensindex = (int *)malloc((pane.panecount + 1) * sizeof(int));
        hipMallocManaged(&pane.lensindex, (pane.panecount + 1) * sizeof(int));
        n = fread(pane.lensindex, 4, pane.panecount + 1, fi);
        // pane.lensdata = (lens_t *)malloc(pane.lensindex[pane.panecount] * sizeof(lens_t));
        hipMallocManaged(&pane.lensdata, pane.lensindex[pane.panecount] * sizeof(lens_t));
        n = fread(pane.lensdata, 16, pane.lensindex[pane.panecount], fi);
        fclose(fi);
    }
    printf("%lu\n", n);
    return 0;
}

__global__ void causcal_kernel(pane_t pane, result_t result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float rs = 1.0f / result.raydensity;
    long long tot = 1ll * result.raydensity * result.raydensity;
    for (long long step=index;step<tot;step+=stride) {
        int i = step / result.raydensity;
        int j = step % result.raydensity;
        float x, y, z, kx, ky, invkz;
        y = (i + 0.5f) * rs;
        x = (j + 0.5f) * rs;
        z = 0.0f;
        kx = 0.0f;
        ky = 0.0f;
        invkz = 1.0f;
        for (int k = 0; k < pane.panecount; k++) {
            x += (pane.panepos[k] - z) * kx * invkz;
            y += (pane.panepos[k] - z) * ky * invkz;
            if (x <= 0.0f || x >= 1.0f || y <= 0.0f || y >= 1.0f)
            {
                goto next;
            }
            z = pane.panepos[k];

            float gx = 0.0f, gy = 0.0f;
            float rx, ry;
            int is = pane.lensindex[k];
            int ie = pane.lensindex[k + 1];
            for (int l = is; l < ie; l++)
            {
                rx = x - pane.lensdata[l].x;
                ry = y - pane.lensdata[l].y;
                float r = rx * rx + ry * ry;
                float invr02 = pane.lensdata[l].r * pane.lensdata[l].r;
                if (r < invr02 * 0.99999f)
                {
                    invr02 = 1.0f / invr02;
                    r = r * invr02;
                    r = 4.0f / (1.0f - r);
                    r = -0.5f * exp(4.0f - r) * r * r * invr02 * pane.lensdata[l].d;
                    gx += r * rx;
                    gy += r * ry;
                }
            }
            kx += gx;
            ky += gy;
            float kp = kx * kx + ky * ky;
            if (kp >= 1.0f) {
                goto next;
            }
            invkz = 1.0f / sqrt(1.0f - kp);
        }
        x += (1.0f - z) * kx * invkz;
        y += (1.0f - z) * ky * invkz;
        if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {
            int pixelindex = int(x * result.sensordensity) + result.sensordensity * int(y * result.sensordensity);
            atomicAdd(&result.sensordata[pixelindex], 1);
        }
        next:;
    }
}

const int nr_blocks = 4096;
const int nr_threads = 512;

int causcal(pane_t pane, result_t result) {
    causcal_kernel<<<nr_blocks, nr_threads>>>(pane, result);
    hipDeviceSynchronize();
    return 0;
}
int main() {
    result_t result;
    pane_t pane;
    loadconf("./conf.data", pane, result);
    causcal(pane, result);
    FILE *fi;
    if (fi = fopen("./out.data", "wb")) {
        fwrite(result.sensordata, 1, result.sensordensity * result.sensordensity * sizeof(int), fi);
        fclose(fi);
    }
    return 0;
}