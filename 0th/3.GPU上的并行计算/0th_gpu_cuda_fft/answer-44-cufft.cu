
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <complex>

void my_fft(int c, int r, std::complex<double> *in, std::complex<double> *out) {
    hipfftHandle plan;
    hipfftPlan3d(&plan, c, c, c, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex*>(in),
                 reinterpret_cast<hipfftDoubleComplex*>(out),
                 HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}